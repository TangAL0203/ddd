#include <thread>
#include "undistort_cuda.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "remap.cuh"

using namespace cv;
using namespace std;

void UndistortImagesCuda::getUndistortMap()
{
    cv::Size imageSize(imgWidth, imgHeight);
    auto t1 = std::chrono::system_clock::now();
    cv::fisheye::initUndistortRectifyMap(cameraMatrix, distCoeffs, Matx33d::eye(), cameraMatrix, imageSize, CV_32FC1, map1, map2);
    auto t2 = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time = t2 - t1;
    cout << "getUndistortMap cost time is: " << elapsed_time.count() * 1000 << "(ms)" << endl;
}

void UndistortImagesCuda::undistortImage(string imagePath)
{
    cout << " Undistorting image... " << endl;

    //    string imagePath;
    //    getline(cin, imagePath);

    Mat inputImage = imread(imagePath, CV_LOAD_IMAGE_COLOR);
    if (!inputImage.data)
    {
        cout << " Could not open or find the image: " << imagePath << endl;
        cout << " Verify if the input images path are absolute," << endl;
        cout << " or change the program directory." << endl;
        exit(EXIT_FAILURE);
    }
    imwrite("src.jpg", inputImage);

    cv::Mat outputImage = cv::Mat(imgHeight, imgWidth, CV_8UC3);

    hipError_t err;
    dim3 block(16, 16);
    dim3 grid((imgWidth + block.x - 1) / block.x, (imgHeight + block.y - 1) / block.y);

    uchar *pSrcImgData = NULL;
    uchar *pDstImgData = NULL;
    float *pMapxData = NULL;
    float *pMapyData = NULL;

    // cout << "src map1 行数: " << map1.rows << endl;
    // cout << "src map1 列数: " << map1.cols << endl;
    // cout << "src 通道: " << map1.channels() << endl;

    map1 = map1.reshape(imgHeight * imgWidth, 1);
    map2 = map2.reshape(imgHeight * imgWidth, 1);

    // hipEvent_t start;
    // hipEvent_t stop;
    // hipEventRecord(start, 0);

    {
        err = hipMalloc(&pMapxData, imgHeight * imgWidth * sizeof(float));
        err = hipMalloc(&pMapyData, imgHeight * imgWidth * sizeof(float));
        err = hipMalloc(&pDstImgData, imgHeight * imgWidth * sizeof(uchar) * channels);
        err = hipMalloc(&pSrcImgData, imgHeight * imgWidth * sizeof(uchar) * channels);
    }
    {
        err = hipMemcpy(pMapxData, map1.data, imgHeight * imgWidth * sizeof(float), hipMemcpyHostToDevice);
        err = hipMemcpy(pMapyData, map2.data, imgHeight * imgWidth * sizeof(float), hipMemcpyHostToDevice);
        err = hipMemcpy(pSrcImgData, inputImage.data, imgHeight * imgWidth * sizeof(uchar) * channels, hipMemcpyHostToDevice);
    }

    auto t1 = std::chrono::system_clock::now();
    // cudaRemap<<<grid, block>>>(pSrcImgData, pDstImgData, pMapxData, pMapyData, imgWidth, imgHeight, imgWidth, imgHeight, channels);
    cudaRemap<<<8, 1024>>>(pSrcImgData, pDstImgData, pMapxData, pMapyData, imgWidth, imgHeight, imgWidth, imgHeight, channels);
    hipDeviceSynchronize();

    auto t2 = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_time = t2 - t1;
    cout << "undistortImage cost time is: " << elapsed_time.count() * 1000 << "(ms)" << endl;

    err = hipGetLastError();
    err = hipMemcpy(outputImage.data, pDstImgData, imgWidth * imgHeight * sizeof(uchar) * channels, hipMemcpyDeviceToHost);

    // float elapsed_time;
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsed_time, start, stop);
    // cout << "undistortImage cost time is: " << elapsed_time*1000 << "(ms)" << endl;

    imwrite("undist.jpg", outputImage);

    cout << " Done !" << endl;
}
