#include "hip/hip_runtime.h"
#include <thread>
#include <stdlib.h>
#include <iostream>
#include "remap.cuh"
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;


__global__ void cudaRemap(uchar *pSrcImg, uchar *pDstImg, float *pMapx, float *pMapy,
                          int inWidth, int inHeight, int outWidth, int outHeight, int channels)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t output_img_size = outWidth * outHeight;
    uint32_t total_thread_number = gridDim.x * blockDim.x;

    for (; idx < output_img_size; idx += total_thread_number) {

        uint32_t idx_y = idx / outWidth;
        uint32_t idx_x = idx - idx_y * outWidth;

        float u = pMapx[idx];
        float v = pMapy[idx];

        int u1 = floor(u);
        int v1 = floor(v);
        int u2 = u1 + 1;
        int v2 = v1 + 1;
        if (u1 >= 0 && v1 >= 0 && u2 < inWidth && v2 < inHeight)
        {
            float dx = u - u1;
            float dy = v - v1;
            float weight1 = (1 - dx) * (1 - dy);
            float weight2 = dx * (1 - dy);
            float weight3 = (1 - dx) * dy;
            float weight4 = dx * dy;

            int resultIdx = idx * 3;
            for (int chan = 0; chan < channels; chan++)
            {
                pDstImg[resultIdx + chan] = uchar(weight1 * pSrcImg[(v1 * inWidth + u1) * 3 + chan] +
                                                  weight2 * pSrcImg[(v1 * inWidth + u2) * 3 + chan] +
                                                  weight3 * pSrcImg[(v2 * inWidth + u1) * 3 + chan] +
                                                  weight4 * pSrcImg[(v2 * inWidth + u2) * 3 + chan] + 0.5);
            }
        }

    }
}